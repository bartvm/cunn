
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/MultiscaleLSTM.cu"
#else

#include "../common.h"

void THNN_(MultiscaleLSTM_updateOutput)(
          THCState *state,
          // Inputs
          THCTensor *input,
          THCudaIntTensor *targets,
          THCudaIntTensor *batches,
          THCudaIntTensor *origins,
          // Inputs
          THCTensor *output,
          THCTensor *cellOutput,
          // Parameters
          THCTensor *inputWeight,
          THCTensor *recurrentWeight,
          THCTensor *bias,
          // Buffers
          THCudaIntTensor *numOutArcs, // Per time step
          THCTensor *normalizingConstants,  // Incoming arcs per step and batch
          THCTensor *xW,
          THCTensor *hR,
          THCTensor *gates,
          THCTensor *outputGates,
          // Config
          int batchSize)
{
  // Get sizes
  int totalInputs = THCudaIntTensor_size(state, targets, 0);
  int inputSize = THCTensor_(size)(state, input, 1);
  int hiddenSize = THCTensor_(size)(state, recurrentWeight, 0);

  // The sequence length is the number of hidden states, excluding the initial
  // This means it's equal to the number of elements in the sequence
  thrust::device_ptr<int> targets_ptr(THCudaIntTensor_data(state, targets));
  int seqLength = *thrust::max_element(targets_ptr, targets_ptr + totalInputs);

  // Resize outputs
  // NOTE They are one longer than the sequence to hold the initial state
  THCTensor_(resize3d)(state, output, seqLength + 1, batchSize, hiddenSize);
  THCTensor_(resize3d)(state, cellOutput, seqLength + 1, batchSize, hiddenSize);

  // Resize buffers
  THCTensor_(resize2d)(state, xW, totalInputs, 4 * hiddenSize);
  THCTensor_(resize2d)(state, gates, totalInputs, 4 * hiddenSize);

  THCTensor_(resize3d)(state, hR, seqLength, batchSize, 4 * hiddenSize);
  THCTensor_(resize3d)(state, outputGates, seqLength, batchSize, hiddenSize);

  THCTensor_(resize2d)(state, normalizingConstants, seqLength, batchSize);
  THCudaIntTensor_resize1d(state, numOutArcs, seqLength);

  // Set cellOutput to zero but leave initial state alone
  THCTensor* cellOutput_ = THCTensor_(newNarrow)(state, cellOutput, 0, 1, seqLength);
  THCTensor_(zero)(state, cellOutput_);

  // Accumulation tensors need to be set to 0 too
  THCTensor_(zero)(state, outputGates);
  THCTensor_(zero)(state, normalizingConstants);
  THCudaIntTensor_zero(state, numOutArcs);

  int nThreads = totalInputs;

  // Count the number of arcs going in and out at each step
  countArcs<real><<<GET_BLOCKS(nThreads), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
    batchSize, totalInputs,
    THCudaIntTensor_data(state, targets),
    THCudaIntTensor_data(state, batches),
    THCudaIntTensor_data(state, origins),
    THCudaIntTensor_data(state, numOutArcs),
    THCTensor_(data)(state, normalizingConstants)
  );

  // Transform the input data
  #ifdef THC_REAL_IS_FLOAT
  THCudaBlas_Sgemm(
  #elif defined(THC_REAL_IS_HALF)
  THCudaBlas_Hgemm(
  #elif defined(THC_REAL_IS_DOUBLE)
  THCudaBlas_Dgemm(
  #endif
    state,
    'n', 'n',
    4 * hiddenSize,
    totalInputs,
    inputSize,
    ScalarConvert<int, real>::to(1),
    THCTensor_(data)(state, inputWeight),
    4 * hiddenSize,
    THCTensor_(data)(state, input),
    inputSize,
    ScalarConvert<int, real>::to(0),
    THCTensor_(data)(state, xW),
    4 * hiddenSize
  );

  // Create tensors to hold the slices at each step
  THCTensor *output_t = THCTensor_(new)(state);
  THCTensor *hR_t = THCTensor_(new)(state);
  THCTensor *xW_t = THCTensor_(new)(state);
  THCTensor *gates_t = THCTensor_(new)(state);
  THCudaIntTensor *targets_t = THCudaIntTensor_new(state);
  THCudaIntTensor *batches_t = THCudaIntTensor_new(state);
  THCTensor *cellOutput_t = THCTensor_(new)(state);
  THCTensor *outputGates_t = THCTensor_(new)(state);
  THCTensor *normalizingConstants_t = THCTensor_(new)(state);

  int inputsSeen = 0;
  for (int t = 0; t < seqLength; t++) {
    // The number of arcs we must process at this time step
    int numOutArcs_t = THCudaIntTensor_get1d(state, numOutArcs, t);

    if (numOutArcs_t != 0) {
      // Transform the hidden state
      THCTensor_(select)(state, output_t, output, 0, t);
      THCTensor_(select)(state, hR_t, hR, 0, t);
      #ifdef THC_REAL_IS_FLOAT
      THCudaBlas_Sgemm(
      #elif defined(THC_REAL_IS_HALF)
      THCudaBlas_Hgemm(
      #elif defined(THC_REAL_IS_DOUBLE)
      THCudaBlas_Dgemm(
      #endif
        state,
        'n', 'n',
        hiddenSize * 4,
        batchSize,
        hiddenSize,
        ScalarConvert<int, real>::to(1),
        THCTensor_(data)(state, recurrentWeight),
        hiddenSize * 4,
        THCTensor_(data)(state, output_t),
        hiddenSize,
        ScalarConvert<int, real>::to(0),
        THCTensor_(data)(state, hR_t),
        hiddenSize * 4
      );

      // Perform the LSTM transitions
      THCTensor_(narrow)(state, xW_t, xW, 0, inputsSeen, numOutArcs_t);
      THCTensor_(narrow)(state, gates_t, gates, 0, inputsSeen, numOutArcs_t);
      THCudaIntTensor_narrow(state, targets_t, targets, 0, inputsSeen, numOutArcs_t);
      THCudaIntTensor_narrow(state, batches_t, batches, 0, inputsSeen, numOutArcs_t);

      inputsSeen += numOutArcs_t;

      nThreads = numOutArcs_t * hiddenSize;

      lstmElemwise<real><<<GET_BLOCKS(nThreads), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
          t, hiddenSize, batchSize,
          THCudaIntTensor_data(state, targets_t),
          THCudaIntTensor_data(state, batches_t),
          numOutArcs_t,
          THCTensor_(data)(state, hR_t),
          THCTensor_(data)(state, xW_t),
          THCTensor_(data)(state, bias),
          THCTensor_(data)(state, gates_t),
          THCTensor_(data)(state, cellOutput),
          THCTensor_(data)(state, outputGates)
      );
    }

    // Average the states of the next time step
    THCTensor_(select)(state, cellOutput_t, cellOutput, 0, t + 1);
    THCTensor_(select)(state, outputGates_t, outputGates, 0, t);
    THCTensor_(select)(state, normalizingConstants_t, normalizingConstants, 0, t);
    THCTensor_(select)(state, output_t, output, 0, t + 1);

    nThreads = batchSize * hiddenSize;

    calculateState<<<GET_BLOCKS(nThreads), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
      hiddenSize, batchSize,
      THCTensor_(data)(state, cellOutput_t),
      THCTensor_(data)(state, outputGates_t),
      THCTensor_(data)(state, normalizingConstants_t),
      THCTensor_(data)(state, output_t)
    );
  }

}

void THNN_(MultiscaleLSTM_backward)(
          THCState *state,
          // Inputs
          THCTensor *input,
          THCTensor *gradInput,
          THCudaIntTensor *targets,
          THCudaIntTensor *batches,
          THCudaIntTensor *origins,
          // Inputs
          THCTensor *output,
          THCTensor *gradOutput,
          THCTensor *cellOutput,
          THCTensor *gradCellOutput,
          // Parameters
          THCTensor *inputWeight,
          THCTensor *gradInputWeight,
          THCTensor *recurrentWeight,
          THCTensor *gradRecurrentWeight,
          THCTensor *bias,
          THCTensor *gradBias,
          // Buffers
          THCudaIntTensor *numOutArcs,
          THCTensor *normalizingConstants,  // Incoming arcs per step and batch
          THCTensor *xW,
          THCTensor *hR,
          THCTensor *gradHR,
          THCTensor *gates,
          THCTensor *gradGates,
          THCTensor *outputGates,
          THCTensor *gradOutputGates,
          int batchSize,
          float scale)
{
  // Get sizes
  int seqLength = THCTensor_(size)(state, output, 0) - 1;
  int totalInputs = THCTensor_(size)(state, input, 0);
  int inputSize = THCTensor_(size)(state, input, 1);
  int hiddenSize = THCTensor_(size)(state, recurrentWeight, 0);

  // Resize buffers
  THCTensor_(resizeAs)(state, gradHR, hR);
  THCTensor_(resizeAs)(state, gradInput, input);
  THCTensor_(resizeAs)(state, gradGates, gates);
  THCTensor_(resizeAs)(state, gradCellOutput, cellOutput);
  THCTensor_(resizeAs)(state, gradOutputGates, outputGates);

  // Accumulation tensors set to zero
  THCTensor_(zero)(state, gradHR);
  THCTensor_(zero)(state, gradGates);
  THCTensor_(zero)(state, gradCellOutput);

  // Create tensors to view slices
  THCTensor *gradOutput_t = THCTensor_(new)(state);
  THCTensor *gradGates_t = THCTensor_(new)(state);
  THCTensor *gradCellOutput_t = THCTensor_(new)(state);
  THCTensor *gradOutputGates_t = THCTensor_(new)(state);

  // Create tensors to hold the slices at each step
  THCTensor *output_t = THCTensor_(new)(state);
  THCTensor *hR_t = THCTensor_(new)(state);
  THCTensor *gradHR_t = THCTensor_(new)(state);
  THCTensor *xW_t = THCTensor_(new)(state);
  THCTensor *gates_t = THCTensor_(new)(state);
  THCudaIntTensor *targets_t = THCudaIntTensor_new(state);
  THCudaIntTensor *batches_t = THCudaIntTensor_new(state);
  THCTensor *cellOutput_t = THCTensor_(new)(state);
  THCTensor *outputGates_t = THCTensor_(new)(state);
  THCTensor *normalizingConstants_t = THCTensor_(new)(state);

  int nThreads;

  int inputsSeen = 0;
  for (int t = seqLength - 1; t >= 0; t--) {
    THCTensor_(select)(state, cellOutput_t, cellOutput, 0, t + 1);
    THCTensor_(select)(state, gradCellOutput_t, gradCellOutput, 0, t + 1);
    THCTensor_(select)(state, outputGates_t, outputGates, 0, t);
    THCTensor_(select)(state, gradOutputGates_t, gradOutputGates, 0, t);
    THCTensor_(select)(state, gradOutput_t, gradOutput, 0, t + 1);
    THCTensor_(select)(state, normalizingConstants_t, normalizingConstants, 0, t);

    nThreads = batchSize * hiddenSize;

    calculateGradState<real><<<GET_BLOCKS(nThreads), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
      hiddenSize, batchSize,
      THCTensor_(data)(state, cellOutput_t),
      THCTensor_(data)(state, gradCellOutput_t),
      THCTensor_(data)(state, outputGates_t),
      THCTensor_(data)(state, gradOutputGates_t),
      THCTensor_(data)(state, normalizingConstants_t),
      THCTensor_(data)(state, gradOutput_t)
    );

    int numOutArcs_t = THCudaIntTensor_get1d(state, numOutArcs, t);
    if (numOutArcs_t == 0) {
      continue;
    }
    inputsSeen += numOutArcs_t;

    THCTensor_(select)(state, hR_t, hR, 0, t);
    THCTensor_(select)(state, gradHR_t, gradHR, 0, t);
    THCTensor_(narrow)(state, xW_t, xW, 0, totalInputs - inputsSeen, numOutArcs_t);
    THCTensor_(narrow)(state, gates_t, gates, 0, totalInputs - inputsSeen, numOutArcs_t);
    THCTensor_(narrow)(state, gradGates_t, gradGates, 0, totalInputs - inputsSeen, numOutArcs_t);
    THCudaIntTensor_narrow(state, targets_t, targets, 0, totalInputs - inputsSeen, numOutArcs_t);
    THCudaIntTensor_narrow(state, batches_t, batches, 0, totalInputs - inputsSeen, numOutArcs_t);

    nThreads = numOutArcs_t * hiddenSize;

    gradLstmElemwise<real><<<GET_BLOCKS(nThreads), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
        t, hiddenSize, batchSize,
        THCudaIntTensor_data(state, targets_t),
        THCudaIntTensor_data(state, batches_t),
        numOutArcs_t,
        THCTensor_(data)(state, hR_t),
        THCTensor_(data)(state, gradHR_t),
        THCTensor_(data)(state, xW_t),
        THCTensor_(data)(state, bias),
        THCTensor_(data)(state, gates_t),
        THCTensor_(data)(state, gradGates_t),
        THCTensor_(data)(state, cellOutput),
        THCTensor_(data)(state, gradCellOutput),
        THCTensor_(data)(state, outputGates),
        THCTensor_(data)(state, gradOutputGates)
    );

    // TODO Separate streams or batched GEMM
    THCState_setCurrentStreamIndex(state, 1);
    cublasSetStream(THCState_getCurrentBlasHandle(state), THCState_getCurrentStream(state));
    THCTensor_(select)(state, output_t, output, 0, t);
    #ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemm(
    #elif defined(THC_REAL_IS_HALF)
    THCudaBlas_Hgemm(
    #elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemm(
    #endif
      state,
      'n', 't',
      hiddenSize * 4,
      hiddenSize,
      batchSize,
      ScalarConvert<float, real>::to(scale),
      THCTensor_(data)(state, gradHR_t),
      hiddenSize * 4,
      THCTensor_(data)(state, output_t),
      hiddenSize,
      ScalarConvert<int, real>::to(1),
      THCTensor_(data)(state, gradRecurrentWeight),
      hiddenSize * 4
    );

    THCState_setCurrentStreamIndex(state, 2);
    cublasSetStream(THCState_getCurrentBlasHandle(state), THCState_getCurrentStream(state));
    THCTensor_(select)(state, gradOutput_t, gradOutput, 0, t);
    #ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemm(
    #elif defined(THC_REAL_IS_HALF)
    THCudaBlas_Hgemm(
    #elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemm(
    #endif
      state,
      't', 'n',
      hiddenSize,
      batchSize,
      hiddenSize * 4,
      ScalarConvert<int, real>::to(1),
      THCTensor_(data)(state, recurrentWeight),
      hiddenSize * 4,
      THCTensor_(data)(state, gradHR_t),
      hiddenSize * 4,
      ScalarConvert<int, real>::to(1),
      THCTensor_(data)(state, gradOutput_t),
      hiddenSize
    );

    THCState_setCurrentStreamIndex(state, 0);

  }

  THCState_setCurrentStreamIndex(state, 1);
  cublasSetStream(THCState_getCurrentBlasHandle(state), THCState_getCurrentStream(state));
  #ifdef THC_REAL_IS_FLOAT
  THCudaBlas_Sgemm(
  #elif defined(THC_REAL_IS_HALF)
  THCudaBlas_Hgemm(
  #elif defined(THC_REAL_IS_DOUBLE)
  THCudaBlas_Dgemm(
  #endif
    state,
    'n', 't',
    4 * hiddenSize,
    inputSize,
    totalInputs,
    ScalarConvert<float, real>::to(scale),
    THCTensor_(data)(state, gradGates),
    4 * hiddenSize,
    THCTensor_(data)(state, input),
    inputSize,
    ScalarConvert<int, real>::to(0),
    THCTensor_(data)(state, gradInputWeight),
    4 * hiddenSize
  );

  THCState_setCurrentStreamIndex(state, 2);
  cublasSetStream(THCState_getCurrentBlasHandle(state), THCState_getCurrentStream(state));
  #ifdef THC_REAL_IS_FLOAT
  THCudaBlas_Sgemm(
  #elif defined(THC_REAL_IS_HALF)
  THCudaBlas_Hgemm(
  #elif defined(THC_REAL_IS_DOUBLE)
  THCudaBlas_Dgemm(
  #endif
    state,
    't', 'n',
    inputSize,
    totalInputs,
    4 * hiddenSize,
    ScalarConvert<int, real>::to(1),
    THCTensor_(data)(state, inputWeight),
    4 * hiddenSize,
    THCTensor_(data)(state, gradGates),
    4 * hiddenSize,
    ScalarConvert<int, real>::to(0),
    THCTensor_(data)(state, gradInput),
    inputSize
  );

  THCState_setCurrentStreamIndex(state, 0);

  THCTensor_(sum)(state, gradBias, gradGates, 0);
  THCTensor_(squeeze1d)(state, gradBias, gradBias, 0);
}

void THNN_(MultiscaleCriterion_updateOutput)(
    // Inputs
    THCState *state,
    THCTensor *input,
    THCudaIntTensor *targets,
    THCudaIntTensor *batches,
    THCudaIntTensor *origins,
    THCudaIntTensor *arcs,
    // Output
    THCTensor *output,
    // Buffers
    THCTensor *stateProbs,
    THCTensor *gradStateProbs,
    THCudaIntTensor *numOutArcs, // Per time step
    THCudaIntTensor *seqLengths,
    bool ignoreLast)
{
  int totalInputs = THCudaIntTensor_size(state, targets, 0);
  int seqLength = THCTensor_(size)(state, input, 0) - (ignoreLast ? 1 : 0);
  int batchSize = THCTensor_(size)(state, input, 1);
  int dictSize = THCTensor_(size)(state, input, 2);

  // Resize buffers and output
  THCudaIntTensor_resize1d(state, seqLengths, batchSize);
  THCTensor_(resize2d)(state, stateProbs, seqLength + 1, batchSize);
  THCudaIntTensor_resize1d(state, numOutArcs, seqLength);
  THCTensor_(resize1d)(state, output, 1);

  // Set accumlating tensors to zero
  THCudaIntTensor_zero(state, numOutArcs);
  THCudaIntTensor_zero(state, seqLengths);
  THCTensor_(zero)(state, output);

  // Initial state probabilities are 1 (so 0 in log-space)
  THCTensor_(fill)(state, stateProbs, THCNumerics<real>::min());
  THCTensor_(fill)(state, THCTensor_(newSelect)(state, stateProbs, 0, 0), ScalarConvert<float, real>::to(0));
  THCudaCheck(cudaDeviceSynchronize());

  // Find the sequence lengths of each example in batch as well as the number of out arcs
  int nThreads = totalInputs;
  findSeqLengths<<<GET_BLOCKS(nThreads), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
    totalInputs,
    THCudaIntTensor_data(state, targets),
    THCudaIntTensor_data(state, batches),
    THCudaIntTensor_data(state, origins),
    THCudaIntTensor_data(state, seqLengths),
    THCudaIntTensor_data(state, numOutArcs)
  );

  THCudaIntTensor *targets_t = THCudaIntTensor_new(state);
  THCudaIntTensor *batches_t = THCudaIntTensor_new(state);
  THCudaIntTensor *origins_t = THCudaIntTensor_new(state);
  THCudaIntTensor *arcs_t = THCudaIntTensor_new(state);
  THCTensor *input_t= THCTensor_(new)(state);

  // Calculate the actual state probabilities
  int inputsSeen = 0;
  for (int t = 0; t < seqLength; t++) {
    int numOutArcs_t = THCudaIntTensor_get1d(state, numOutArcs, t);

    THCudaIntTensor_narrow(state, targets_t, targets, 0, inputsSeen, numOutArcs_t);
    THCudaIntTensor_narrow(state, batches_t, batches, 0, inputsSeen, numOutArcs_t);
    THCudaIntTensor_narrow(state, origins_t, origins, 0, inputsSeen, numOutArcs_t);
    THCudaIntTensor_narrow(state, arcs_t, arcs, 0, inputsSeen, numOutArcs_t);
    THCTensor_(select)(state, input_t, input, 0, t);

    inputsSeen += numOutArcs_t;

    nThreads = numOutArcs_t;
    if (numOutArcs_t != 0) {
      calculateStateProbs<real><<<GET_BLOCKS(nThreads), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
        batchSize, dictSize, numOutArcs_t,
        THCTensor_(data)(state, input_t),
        THCTensor_(data)(state, stateProbs),
        THCudaIntTensor_data(state, targets_t),
        THCudaIntTensor_data(state, batches_t),
        THCudaIntTensor_data(state, origins_t),
        THCudaIntTensor_data(state, arcs_t)
      );
    }
  }

  // We set the gradients to 1 already
  THCTensor_(resizeAs)(state, gradStateProbs, stateProbs);
  THCTensor_(zero)(state, gradStateProbs);

  nThreads = batchSize;
  sumStateProbs<real><<<GET_BLOCKS(nThreads), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
    batchSize,
    THCTensor_(data)(state, stateProbs),
    THCudaIntTensor_data(state, seqLengths),
    THCTensor_(data)(state, output),
    THCTensor_(data)(state, gradStateProbs)
  );

  // The cost can be divided by batchSize, the sum of sequence lengths, or totalInputs
  // THCTensor_(div)(state, output, output, ScalarConvert<float, real>::to(totalInputs));
}

void THNN_(MultiscaleCriterion_updateGradInput)(
    // Inputs
    THCState *state,
    THCTensor *input,
    THCTensor *gradInput,
    THCudaIntTensor *targets,
    THCudaIntTensor *batches,
    THCudaIntTensor *origins,
    THCudaIntTensor *arcs,
    // Output
    THCTensor *output,
    // Buffers
    THCTensor *stateProbs,
    THCTensor *gradStateProbs,
    THCudaIntTensor *numOutArcs, // Per time step
    THCudaIntTensor *seqLengths,
    bool ignoreLast)
{
  int totalInputs = THCudaIntTensor_size(state, targets, 0);
  int seqLength = THCTensor_(size)(state, input, 0) - (ignoreLast ? 1 : 0);
  int batchSize = THCTensor_(size)(state, input, 1);
  int dictSize = THCTensor_(size)(state, input, 2);

  THCTensor_(resizeAs)(state, gradInput, input);
  THCTensor_(zero)(state, gradInput);

  THCudaIntTensor *targets_t = THCudaIntTensor_new(state);
  THCudaIntTensor *batches_t = THCudaIntTensor_new(state);
  THCudaIntTensor *origins_t = THCudaIntTensor_new(state);
  THCudaIntTensor *arcs_t = THCudaIntTensor_new(state);
  THCTensor *input_t= THCTensor_(new)(state);
  THCTensor *gradInput_t= THCTensor_(new)(state);

  int nThreads;

  int inputsSeen = 0;
  for (int t = seqLength - 1; t >= 0; t--) {
    int numOutArcs_t = THCudaIntTensor_get1d(state, numOutArcs, t);
    inputsSeen += numOutArcs_t;

    THCudaIntTensor_narrow(state, targets_t, targets, 0, totalInputs - inputsSeen, numOutArcs_t);
    THCudaIntTensor_narrow(state, batches_t, batches, 0, totalInputs - inputsSeen, numOutArcs_t);
    THCudaIntTensor_narrow(state, origins_t, origins, 0, totalInputs - inputsSeen, numOutArcs_t);
    THCudaIntTensor_narrow(state, arcs_t, arcs, 0, totalInputs - inputsSeen, numOutArcs_t);
    THCTensor_(select)(state, input_t, input, 0, t);
    THCTensor_(select)(state, gradInput_t, gradInput, 0, t);

    nThreads = numOutArcs_t;
    calculateGradStateProbs<real><<<GET_BLOCKS(nThreads), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
      batchSize, dictSize, numOutArcs_t,
      THCTensor_(data)(state, input_t),
      THCTensor_(data)(state, gradInput_t),
      THCTensor_(data)(state, stateProbs),
      THCTensor_(data)(state, gradStateProbs),
      THCudaIntTensor_data(state, targets_t),
      THCudaIntTensor_data(state, batches_t),
      THCudaIntTensor_data(state, origins_t),
      THCudaIntTensor_data(state, arcs_t)
    );

  }

}
#endif
